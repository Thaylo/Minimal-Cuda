// montecarlo_kernels.cu
#include <cstdio>
#include <cmath>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call) do {                                \
    hipError_t err = (call);                                \
    if (err != hipSuccess) {                                \
        fprintf(stderr, "CUDA Error at %s:%d: %s\n",         \
                __FILE__, __LINE__, hipGetErrorString(err));\
        exit(EXIT_FAILURE);                                  \
    }                                                        \
} while (0)

__global__ void initRNGKernel(hiprandState* states, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void monteCarloKernel(hiprandState* states,
                                 float* d_payoffs,
                                 float  S0, 
                                 float  K,
                                 float  r, 
                                 float  sigma,
                                 float  T,
                                 int    N) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float z = hiprand_normal(&states[idx]);
        float ST = S0 * expf((r - 0.5f * sigma * sigma) * T +
                             sigma * sqrtf(T) * z);
        float payoff = fmaxf(ST - K, 0.0f);
        d_payoffs[idx] = payoff;
    }
}

// Host wrapper function callable from C++ code.
extern "C" void runMonteCarlo(float S0, float K, float r, float sigma, float T,
                              int N, unsigned long seed, float discount, double* optionPrice)
{
    hiprandState* d_states = nullptr;
    float* d_payoffs = nullptr;
    CUDA_CHECK(hipMalloc(&d_states, N * sizeof(hiprandState)));
    CUDA_CHECK(hipMalloc(&d_payoffs, N * sizeof(float)));

    const int BLOCK_SIZE = 256;
    const int GRID_SIZE  = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Initialize RNG on device.
    initRNGKernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_states, seed);
    CUDA_CHECK(hipDeviceSynchronize());
 
    // Launch the Monte Carlo kernel.
    monteCarloKernel<<<GRID_SIZE, BLOCK_SIZE>>>(
        d_states, d_payoffs, 
        S0, K, r, sigma, T, N
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy results back to host.
    float* h_payoffs = (float*)malloc(N * sizeof(float));
    CUDA_CHECK(hipMemcpy(h_payoffs, d_payoffs, N * sizeof(float), hipMemcpyDeviceToHost));

    double sum = 0.0;
    for (int i = 0; i < N; ++i) {
        sum += (double)h_payoffs[i] / N;
    }
    *optionPrice = discount * sum;

    free(h_payoffs);
    CUDA_CHECK(hipFree(d_states));
    CUDA_CHECK(hipFree(d_payoffs));
}
